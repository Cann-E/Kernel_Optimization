#include "hip/hip_runtime.h"

#include <cassert> 
#include <hip/hip_runtime.h>
#include "transpose_device.cuh"
#include <stdio.h>
#include <hip/hip_runtime.h>

/*
 * TODO for all kernels (including naive):
 * Leave a comment above all non-coalesced memory accesses and bank conflicts.
 * Make it clear if the suboptimal access is a read or write. If an access is
 * non-coalesced, specify how many cache lines it touches, and if an access
 * causes bank conflicts, say if its a 2-way bank conflict, 4-way bank
 * conflict, etc.
 *
 * Comment all of your kernels.
 */



#define BS 32
__global__
void naiveTransposeKernel(const float *input, float *output, int n) {
    // TODO: do not modify code, just comment on suboptimal accesses
    // Each thread loads one element from input and writes one element to output.
    // But memory accesses are bad non-coalesced.
    // - Reads: Threads in a warp access different rows → slow global memory access.
    // - Writes: Threads write scattered values in output → bad for memory efficiency.

    int i = threadIdx.x + blockDim.x * blockIdx.x;
    int j = threadIdx.y + blockDim.y * blockIdx.y;


    output[j + n * i] = input[i + n * j];

}


__global__
void shmemTransposeKernel(const float *input, float *output, int n) {
    // TODO: Modify transpose kernel to use shared memory. 
    // All global memory reads and writes should be coalesced. 
    // Minimize the number of shared memory bank conflicts 
    // (0 bank conflicts should be possible using padding).
    
    __shared__ float tile[BS][BS + 1]; // Adding padding to avoid bank conflicts

    int x = threadIdx.x;
    int y = threadIdx.y;
    int i = x + blockDim.x * blockIdx.x;
    int j = y + blockDim.y * blockIdx.y;

    // Load data into shared memory
    if (i < n && j < n) {
        tile[y][x] = input[j * n + i]; 
    }

    __syncthreads(); // Sync before writing back

    // Transpose and write back
    int transposed_i = y + blockDim.y * blockIdx.y;
    int transposed_j = x + blockDim.x * blockIdx.x;

    if (transposed_i < n && transposed_j < n) {
        output[transposed_j * n + transposed_i] = tile[y][x];
    }
}



__global__
void optimalTransposeKernel(const float *input, float *output, int n) {
    // TODO: This should be based off of your shmemTransposeKernel.
    // Use any optimization tricks discussed so far to improve performance.
    // Consider ILP and loop unrolling (thread coarsening)

    __shared__ float tile[BS][BS + 1]; // Shared memory to reduce global memory access

    int x = threadIdx.x;
    int y = threadIdx.y;
    int i = x + blockDim.x * blockIdx.x;
    int j = y + blockDim.y * blockIdx.y;

    
    if (i < n && j < n) {
        tile[y][x] = input[j * n + i]; // Coalesced read into shared memory
    }
    __syncthreads();

    // Transpose the indices within the shared memory
    int transposed_i = blockIdx.y * blockDim.y + threadIdx.x;
    int transposed_j = blockIdx.x * blockDim.x + threadIdx.y;

    
    if (transposed_i < n && transposed_j < n) {
        output[transposed_j * n + transposed_i] = tile[x][y];
    }
}




void cudaTranspose(
    const float *d_input,
    float *d_output,
    int n,
    TransposeImplementation type)
{
    // TODO: you can change the block dims

    dim3 blockSize(32, 32);
    dim3 gridSize((n + 31) / 32, (n + 31) / 32);

    if (type == NAIVE) {
        naiveTransposeKernel<<<gridSize, blockSize>>>(d_input, d_output, n);
    }
    else if (type == SHMEM) {
        shmemTransposeKernel<<<gridSize, blockSize>>>(d_input, d_output, n);
    }
    else if (type == OPTIMAL) {
        optimalTransposeKernel<<<gridSize, blockSize>>>(d_input, d_output, n);
    }
    else
        assert(false);
}


